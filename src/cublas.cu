#include <hip/hip_runtime.h>
#include <cblas.h>

#include <hipsolver.h>
#include <hipblas.h>
#include <iostream>
#include <cstring>
#include <cmath>
#define DEBUG 0

using namespace std;

//ToDo: includere utils tramite utils.h
extern void print_array_as_matrix(int*, long, char*);
extern void print_array_as_matrix(float*, long, char*);
extern int* createRandomMatrixArray(long, long, bool);
extern void saveTimeToFile(long, double, char*);

int main(int argc, char **argv){

    long min_dim, max_dim, step, dim, data_size, smaller_size;
    int *S;
    float *A, *B, *C;
    float *gpu_A, *gpu_B, *gpu_C, *gpu_Work;
    int *gpu_pivot , *gpu_info , Lwork;   // pivots , info , worksp. size
    int info_gpu = 0;
    float time1,time2,time3;
    float  alfa=1.0f;
    float  beta=0.0f;
    int  incx=1, incy =1;
    hipError_t status;
    hipEvent_t begin, stop;
    hipblasStatus_t  stat; //CUBLAS functions status
    hipblasHandle_t  handle; //CUBLAS context
    hipsolverStatus_t  cusolverStatus;
    hipsolverHandle_t  cuhandle;
    hipEventCreate(&begin);
    hipEventCreate(&stop);

    if(argc != 4){
        cout << "Usage: " << argv[0] << " [min_dim] [max_dim] [step]" << endl;
        return -1;
    }

    min_dim = strtol(argv[1], NULL, 10);
    max_dim = strtol(argv[2], NULL, 10)+1;
    step = strtol(argv[3], NULL, 10);

    for(dim=min_dim;dim<max_dim;dim+=step){

        //Matrix as a sequential array

        S = createRandomMatrixArray(dim, dim, true); //true means "invertible"
        A = new float[dim*dim];
        for(int i=0;i<dim;i++) for(int j=0;j<dim;j++) A[i*dim+j] = (float)S[i*dim+j];
        free(S);
        S = createRandomMatrixArray(dim, dim, false); //false means "invertible"
        B = new float[dim*dim];
        for(int i=0;i<dim;i++) for(int j=0;j<dim;j++) B[i*dim+j] = (float)S[i*dim+j];
        free(S);
        C = new float[dim*dim];
        for(int i=0;i<dim;i++) for(int j=0;j<dim;j++) C[i*dim+j] = 0;

        data_size = dim*dim*sizeof(float);

        status = hipMalloc((void**) &gpu_A, data_size);
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        status = hipMalloc((void**) &gpu_B, data_size);  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        status = hipMalloc((void**) &gpu_C, data_size);  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        if(DEBUG){
            print_array_as_matrix(A,dim,"A ");
            print_array_as_matrix(B,dim,"B ");
        }

        stat = hipblasCreate(&handle);

        hipEventRecord(begin, 0); //start time measure

        //----------------------CUBLAS CHARGE CODE----------------------

        stat = hipblasSetMatrix(dim,dim,data_size,A,dim,gpu_A,dim);//a -> gpu_A
        stat = hipblasSetMatrix(dim,dim,data_size,B,dim,gpu_B,dim);//b -> gpu_B
        stat = hipblasSetMatrix(dim,dim,data_size,C,dim,gpu_C,dim);//c -> gpu_C

        //----------------------CUBLAS CHARGE CODE----------------------
        hipDeviceSynchronize(); //to reassure everything is in sync
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &time1, begin, stop);

        if(DEBUG) cout << "MUL_GCHR: With dimension " << dim << ", elapsed time: " <<  time1 << " ms" << endl;
        saveTimeToFile(dim, time1/1000, "csv/load_multiplication_CUBLAS.csv");     

        hipEventRecord(begin, 0);
        //----------------------CUBLAS PARALLEL CODE----------------------

        // C := alfa*A*B + beta*C;
        stat=hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,dim,dim,&alfa,gpu_A,dim,gpu_B,dim,&beta,gpu_C,dim);

        //----------------------CUBLAS PARALLEL CODE---------------------- 

        hipDeviceSynchronize(); //to reassure everything is in sync

        hipEventRecord(stop, 0);

        hipEventSynchronize(stop);

        hipEventElapsedTime( &time2, begin, stop);

        if(DEBUG) cout << "MUL_PRLL: With dimension " << dim << ", elapsed time: " << time2 << " ms" << endl;

        hipEventRecord(begin, 0); //start time measure

        //----------------------CUBLAS DISCHARGE CODE----------------------
        //si basa sul fatto che i caricamenti sono sincroni, mentre l'esecuzione parallela no        

        stat=hipblasGetMatrix(dim,dim,data_size,gpu_C,dim,C,dim); // gpu_C -> C

        hipDeviceSynchronize();

        //----------------------CUBLAS DISCHARGE CODE----------------------

        hipDeviceSynchronize(); //to reassure everything is in sync
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &time3, begin, stop);
        

        if(DEBUG) cout << "MUL_CCHR: With dimension " << dim << ", elapsed time: " << time3 << " ms" << endl;
        saveTimeToFile(dim, time3/1000, "csv/read_multiplication_CUBLAS.csv");

        saveTimeToFile(dim, (time1+time2+time3)/1000, "csv/multiplication_CUBLAS.csv");

        if(DEBUG){
            print_array_as_matrix(C,dim,"MULT ");
        }

        hipFree(gpu_A);
        hipFree(gpu_B);
        hipFree(gpu_C);
        free(B);
        free(C);
        hipblasDestroy(handle);   

        B = new float[dim];
        C = new float[dim];

        for(int i=0;i<dim;i++) B[i] = 0.0;                //  initialize B
        for(int i=0;i<dim;i++) C[i] = 1.0;    // C - N-vector  of ones

        cusolverStatus = hipsolverDnCreate (& cuhandle ); 

        status = hipMalloc((void**) &gpu_A, data_size);
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        status = hipMalloc((void**) &gpu_B, dim*sizeof(float));  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        status = hipMalloc((void**) &gpu_pivot, dim*sizeof(int));  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        status = hipMalloc((void**) &gpu_info, dim*sizeof(int));  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        smaller_size = dim*sizeof(float);

        hipEventRecord(begin, 0); //start time measure

        //----------------------CUBLAS CHARGE CODE----------------------
        //si basa sul fatto che i caricamenti sono sincroni, mentre l'esecuzione parallela no        

        status = hipMemcpy(gpu_A, A, data_size,hipMemcpyHostToDevice);      // copy d_A <-A

        //moltiplica B = A*C
        cblas_sgemv(CblasColMajor,CblasNoTrans,dim,dim,alfa,A,dim,C,incx,beta,B,incy);

        status = hipMemcpy(gpu_B, B, smaller_size,hipMemcpyHostToDevice);      // copy d_B <-B

        cusolverStatus = hipsolverDnSgetrf_bufferSize(cuhandle,dim,dim,gpu_A,dim,&Lwork);      //  compute  buffer  size  and  prep.memory

        //----------------------CUBLAS CHARGE CODE----------------------

        hipDeviceSynchronize(); //to reassure everything is in sync
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &time1, begin, stop);

        if(DEBUG) cout << "INV_GCHR: With dimension " << dim << ", elapsed time: " << time1 << " ms" << endl;
        saveTimeToFile(dim, time1/1000, "csv/load_inversion_CUBLAS.csv");

        status = hipMalloc((void**) &gpu_Work, Lwork*sizeof(float));  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        hipEventRecord(begin, 0);

        //----------------------CUBLAS PARALLEL CODE----------------------

        cusolverStatus = hipsolverDnSgetrf(cuhandle,dim,dim,gpu_A,dim,gpu_Work,gpu_pivot,gpu_info);
        cusolverStatus = hipsolverDnSgetrs(cuhandle, HIPBLAS_OP_N,dim,1,gpu_A,dim,gpu_pivot,gpu_B,dim,gpu_info);

        //----------------------CUBLAS PARALLEL CODE----------------------

        hipDeviceSynchronize(); //to reassure everything is in sync

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &time2, begin, stop);

        if(DEBUG) cout << "INV_PRLL: With dimension " << dim << ", elapsed time: " << time2 << " ms" << endl;
        
        hipEventRecord(begin, 0); //start time measure

        //----------------------CUBLAS DISCHARGE CODE----------------------
        //si basa sul fatto che i caricamenti sono sincroni, mentre l'esecuzione parallela no        

        status = hipMemcpy (&info_gpu , gpu_info , sizeof(int), hipMemcpyDeviceToHost );
        if(DEBUG) cout << "after getrf+getrs: info_gpu = " << info_gpu << endl;
        status = hipMemcpy(C, gpu_B , dim*sizeof(float), hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        //----------------------CUBLAS DISCHARGE CODE----------------------

        hipDeviceSynchronize(); //to reassure everything is in sync
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &time3, begin, stop);
         
        if(DEBUG) cout << "INV_CCHR: With dimension " << dim << ", elapsed time: " << time3 << " ms" << endl;
        saveTimeToFile(dim, time3/1000, "csv/read_inversion_CUBLAS.csv"); 
        saveTimeToFile(dim, (time1+time2+time3)/1000, "csv/inversion_CUBLAS.csv");

        if(DEBUG){
            print_array_as_matrix(C,dim,"C ");
        }

        //deallocate things

        hipFree(gpu_A);
        hipFree(gpu_B);
        hipFree(gpu_pivot);
        hipFree(gpu_info);
        hipFree(gpu_Work);
        free(A);
        free(B);
        free(C);
        cusolverStatus = hipsolverDnDestroy(cuhandle);
    }
 
    hipEventDestroy(begin);
    hipEventDestroy(stop);

    return 0;
}
