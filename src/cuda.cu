#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cstring>
#include <cmath>
#define DEBUG 0
//If DEBUG is setted, the program will print the used matrices and the times on the stdout

using namespace std;

/*function marked with '__global__' are the GPU Kernels*/

//This reduces the matrix to upper triangular
 __global__ void upperReduction(float *A,  float *I, int n, int piv){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    float p;
    //this is still the same gauss jordan algorithm used in other files
    if(i<n && j<n) //to ensure we are within the matrix boundaries
        if(i>piv){ // limits operation to rows below the pivot point
            p = A[i*n+piv]/A[piv*n+piv];
            I[i*n+j] -= I[piv*n+j]*p;  // apply for each row member
            if(j>=piv){ //limits to row members to the right of the pivot
                A[i*n+j] -= A[piv*n+j]*p;  // apply only to members right of pivot
            }
        }
 }

//Reduces the matrix to lower triangular matrix
  __global__ void lowerReduction(float *A,  float *I, int n, int piv){

    //same function of before, but row and col are reversed
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    float p;
    if(i<n && j<n)
        if(i<piv){
            p = A[i*n+piv]/A[piv*n+piv];
            I[i*n+j] -= I[piv*n+j]*p;
            if(j<=piv){
                A[i*n+j] -= A[piv*n+j]*p;
            }
        }
 }

//Scales down the matrix in respect of the elements on the diagonal
 __global__ void scale(float *A,  float *I, int h){
    
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(row<h && col<h)//to ensure we are withing not the matrix boundaries
    {
        I[row*h+col]  /= A[row*(h+1)];
        A[row*h+col] /= A[row*(h+1)];
    }
}

__global__ void matrixMultiplication(float* A, float* B, float* C, int n) {

    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    float c=0;

    // each thread computes one element of the block sub-matrix (and therefore one non-overlapping sub-matrix of C)

    if (row<n && col<n) { //to ensure we are not withing the matrix boundaries
        for (int i = 0; i < n; i++) {
            c += A[row*n+i] * B[i*n+col];
        }

        C[row*n+col] = c;
    }
}


/* From utils.cpp */
extern float* createRandomMatrixArray(long, long, bool);
extern float* createIdentityMatrixArray(long);
extern float* createEmptyMatrixArray(long);
extern void print_array_as_matrix(float*, long, char*);
extern void saveTimeToFile(long, float, char*);
extern bool multipliedMatrixCudaIsCorrect(float*, float*, float*, long);


int main(int argc, char **argv){

    long min_dim, max_dim, step, dim, data_size; //Used to determine which matrix dimensions we will test
    float *A, *B, *C; //After moltiplication , C=A*B
    float *D, *M; //M=A, D=Identity and after inversion: D = A^-1, M=Identity
    float *gpu_A, *gpu_B, *gpu_C; //GPU Matrices
    float *gpu_inv_A, *gpu_inv_I;
    float time; //Will contain elapsed time returned by CUDA events, in milliseconds
    chrono::high_resolution_clock::time_point start, finish; //Used to implement time measurement
    chrono::duration<double> elapsed1, elapsed2; //Used to contain the elapsed time  
    hipError_t status; //variable for error handling
    hipEvent_t begin, stop; //used to the time measurement of the functions on the GPU
    hipEventCreate(&begin); //initialize objects
    hipEventCreate(&stop);

    // Print the usage command if too few parameters were passed
    if(argc != 4){
        cout << "Usage: " << argv[0] << " [min_dim] [max_dim] [step]" << endl;
        return -1;
    }

    min_dim = strtol(argv[1], NULL, 10);
    max_dim = strtol(argv[2], NULL, 10)+1; //'+1' means we will evaluate the "max_dim" value passed as a argument
    step = strtol(argv[3], NULL, 10);

    //for each 'dim' from 'min_dim' to 'max_dim', with the step we chosen 
    for(dim=min_dim;dim<max_dim;dim+=step){

        //Matrices are created and used as arrays
        A = createRandomMatrixArray(dim, dim, true); //true means "invertible"
        B = createRandomMatrixArray(dim, dim, false); //true means "not invertible"
        C = createEmptyMatrixArray(dim);

        //Number of bytes contained in one matrix
        data_size = dim*dim*sizeof(float);

        dim3 threadsPerBlock(dim, dim);
        dim3 blocksPerGrid(1, 1);
        if (dim*dim > 512){ //total amount of threads in a single block cannot exceed 1024 (with a maxwell nVidia GPU)
            threadsPerBlock.x = 512; 
            threadsPerBlock.y = 512;
            blocksPerGrid.x = ceil(float(dim)/float(threadsPerBlock.x));
            blocksPerGrid.y = ceil(float(dim)/float(threadsPerBlock.y));
        }

        //allocate memory to contain the matrices
        status = hipMalloc((void**) &gpu_A, data_size);
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        status = hipMalloc((void**) &gpu_B, data_size);  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        status = hipMalloc((void**) &gpu_C, data_size);  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        if(DEBUG){
            print_array_as_matrix(A,dim,"A ");
            print_array_as_matrix(B,dim,"B ");
        }

        //BEGIN MATRICES MULTIPLICATION

        start = chrono::high_resolution_clock::now(); //start time measure

        //----------------------CUDA CHARGE CODE----------------------
        //copy the matrices A and B from RAM to GPU RAM        

        status = hipMemcpy(gpu_A, A, data_size, hipMemcpyHostToDevice);
        status = hipMemcpy(gpu_B, B, data_size, hipMemcpyHostToDevice);

        hipDeviceSynchronize(); //to reassure the copy has ended

        //----------------------CUDA CHARGE CODE----------------------

        finish = chrono::high_resolution_clock::now(); //end time measure
        elapsed1 = finish - start; //compute time difference

        //elapsed.count() gives the time in seconds
        if(DEBUG) cout << "MUL_GCHR: With dimension " << dim << ", elapsed time: " << elapsed1.count() << " s" << endl;
        
        //Save how much time the load took
        saveTimeToFile(dim, elapsed1.count(), "csv/load_multiplication_CUDA.csv");

        hipEventRecord(begin, 0); //begin "recording" operations on GPU
        
        //----------------------CUDA PARALLEL CODE----------------------
        //load and execute the kernel to multiplication into the GPU

        matrixMultiplication <<< blocksPerGrid, threadsPerBlock >>> (gpu_A, gpu_B, gpu_C, dim);

        //----------------------CUDA PARALLEL CODE---------------------- 

        hipDeviceSynchronize(); //GPU kernel calls are asynchronous, so this is necessary

        //Find how much time the GPU spent on computing. 
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &time, begin, stop);        

        if(DEBUG) cout << "MUL_PRLL: With dimension " << dim << ", elapsed time: " << time << " ms" << endl;

        start = chrono::high_resolution_clock::now(); //start time measure

        //----------------------CUDA DISCHARGE CODE----------------------
        //Reading and paste back on RAM the result matrix        

        status = hipMemcpy(C, gpu_C, data_size, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        //----------------------CUDA DISCHARGE CODE----------------------

        finish = chrono::high_resolution_clock::now(); //end time measure

        elapsed2 = finish - start; //compute time difference



       if(DEBUG) cout << "MUL_CCHR: With dimension " << dim << ", elapsed time: " << elapsed2.count() << " s" << endl;
       
       //Save how much time the read of the result took
       saveTimeToFile(dim, elapsed2.count(), "csv/read_multiplication_CUDA.csv");
       
       //Save how much time the whole computation took (load+calculations+read)
       //Note: 'time' is in milliseconds
       saveTimeToFile(dim, elapsed1.count()+elapsed2.count()+time/1000, "csv/multiplication_CUDA.csv");

        if(DEBUG){
            print_array_as_matrix(C,dim,"C ");
            bool correct = multipliedMatrixCudaIsCorrect(A,B,C,dim);
            if(!correct){
                cout << "Multiplied matrix is not correct, aborting..." << endl;
                return -1;
            }
        }


                
        //Free useless memory on the GPU and on the RAM
        hipFree(gpu_A);
        hipFree(gpu_B);
        hipFree(gpu_C);
        free(B);
        free(C);

        //BEGIN MATRIX INVERSION

        D = createIdentityMatrixArray(dim);

        //M=A
        M = new float[dim*dim];
        for (int h = 0; h < dim; h++){
            for (int w = 0; w < dim; w++)
                    M[h*dim+w] = A[h*dim+w];
        }

        //Number of bytes contained in one matrix
        data_size = dim*dim*sizeof(float);

        //allocate memory to contain the matrices
        status = hipMalloc((void**) &gpu_inv_A, data_size);
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        status = hipMalloc((void**) &gpu_inv_I, data_size);  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        start = chrono::high_resolution_clock::now(); //start time measure

        //----------------------CUDA CHARGE CODE----------------------
        
        status = hipMemcpy(gpu_inv_A, M, data_size, hipMemcpyHostToDevice);
        status = hipMemcpy(gpu_inv_I, D, data_size, hipMemcpyHostToDevice);

        hipDeviceSynchronize();

        //----------------------CUDA CHARGE CODE----------------------

        finish = chrono::high_resolution_clock::now(); //end time measure
        elapsed1 = finish - start; //compute time difference
        if(DEBUG) cout << "INV_GCHR: With dimension " << dim << ", elapsed time: " << elapsed1.count() << " s" << endl;
        saveTimeToFile(dim, elapsed1.count(), "csv/load_inversion_CUDA.csv");

        hipEventRecord(begin, 0);

        //----------------------CUDA PARALLEL CODE----------------------
        //the whole 'for' reduces the matrix to diagonal
        //each call computes from a different line of pivot (passed with 'i')
        //NOTE: every kernel call waits for the previous one to finish

        for(int i=0;i<dim-1;i++){ 
            upperReduction <<< blocksPerGrid, threadsPerBlock >>> (gpu_inv_A, gpu_inv_I, dim, i);
        }
        for(int i=dim-1;i>0;i--){ 
            lowerReduction <<< blocksPerGrid, threadsPerBlock >>> (gpu_inv_A, gpu_inv_I, dim, i);
        }

        //this function scales the starting A matrix to the identity, so "I" will be the correct inverse
        scale <<< blocksPerGrid, threadsPerBlock >>> (gpu_inv_A, gpu_inv_I, dim); //reduce matrix to diagonal

        //----------------------CUDA PARALLEL CODE---------------------- 

        hipDeviceSynchronize(); //to reassure everything is in sync

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &time, begin, stop);

        if(DEBUG) cout << "INV_PRLL: With dimension " << dim << ", elapsed time: " << time << " ms" << endl;
      
        start = chrono::high_resolution_clock::now(); //start time measure

        //----------------------CUDA DISCHARGE CODE----------------------
        //Reads back M and D        

        status = hipMemcpy(M, gpu_inv_A, data_size, hipMemcpyDeviceToHost);
        status = hipMemcpy(D, gpu_inv_I, data_size, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        //----------------------CUDA DISCHARGE CODE----------------------

        finish = chrono::high_resolution_clock::now(); //end time measure

        elapsed2 = finish - start; //compute time difference
         
        if(DEBUG) cout << "INV_CCHR: With dimension " << dim << ", elapsed time: " << elapsed2.count() << " s" << endl;
        saveTimeToFile(dim, elapsed2.count(), "csv/read_inversion_CUDA.csv");
        saveTimeToFile(dim, elapsed1.count()+elapsed2.count()+time/1000, "csv/inversion_CUDA.csv");

        if(DEBUG){
            print_array_as_matrix(D,dim,"D ");
            print_array_as_matrix(M,dim,"M ");
            bool correct = multipliedMatrixCudaIsCorrect(A,D,M,dim);
            if(!correct){
                cout << "Multiplied matrix is not correct, aborting..." << endl;
                return -1;
            }
        }

        //deallocate 
        hipFree(gpu_inv_A);
        hipFree(gpu_inv_I);
        free(A);
        free(D);
        free(M);        

    }
 
    hipEventDestroy(begin);
    hipEventDestroy(stop);  

    return 0;
}
