#include <hip/hip_runtime.h>
#include <cblas.h>

#include <hipsolver.h>
#include <hipblas.h>
#include <iostream>
#include <chrono>
#include <cstring>
#include <cmath>
#define DEBUG 0

using namespace std;

//ToDo: includere utils tramite utils.h
extern void print_array_as_matrix(int*, unsigned, char*);
extern void print_array_as_matrix(float*, unsigned, char*);
extern int* createRandomMatrixArray(unsigned, unsigned, bool);

int main(int argc, char **argv){

    long min_dim, max_dim, step, dim, data_size, smaller_size;
    int *S;
    float *A, *B, *C;
    float *gpu_A, *gpu_B, *gpu_C, *gpu_Work;
    int *gpu_pivot , *gpu_info , Lwork;   // pivots , info , worksp. size
    int info_gpu = 0;
    float time;
    float  alfa=1.0f;
    float  beta=0.0f;
    int  incx=1, incy =1;
    chrono::high_resolution_clock::time_point start, finish;
    chrono::duration<double> elapsed; 
    hipError_t status;
    hipEvent_t begin, stop;
    hipblasStatus_t  stat; //CUBLAS functions status
    hipblasHandle_t  handle; //CUBLAS context
    hipsolverStatus_t  cusolverStatus;
    hipsolverHandle_t  cuhandle;
    hipEventCreate(&begin);
    hipEventCreate(&stop);

    if(argc != 4){
        cout << "Usage: " << argv[0] << " [min_dim] [max_dim] [step]" << endl;
        return -1;
    }

    min_dim = strtol(argv[1], NULL, 10);
    max_dim = strtol(argv[2], NULL, 10)+1;
    step = strtol(argv[3], NULL, 10);

    for(dim=min_dim;dim<max_dim;dim+=step){

        //Matrix as a sequential array

        S = createRandomMatrixArray(dim, dim, true); //true means "invertible"
        A = new float[dim*dim];
        for(int i=0;i<dim;i++) for(int j=0;j<dim;j++) A[i*dim+j] = (float)S[i*dim+j];
        free(S);
        S = createRandomMatrixArray(dim, dim, false); //false means "invertible"
        B = new float[dim*dim];
        for(int i=0;i<dim;i++) for(int j=0;j<dim;j++) B[i*dim+j] = (float)S[i*dim+j];
        free(S);
        C = new float[dim*dim];
        for(int i=0;i<dim;i++) for(int j=0;j<dim;j++) C[i*dim+j] = 0;

        data_size = dim*dim*sizeof(float);

        status = hipMalloc((void**) &gpu_A, data_size);
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        status = hipMalloc((void**) &gpu_B, data_size);  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        status = hipMalloc((void**) &gpu_C, data_size);  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        if(DEBUG){
            print_array_as_matrix(A,dim,"A ");
            print_array_as_matrix(B,dim,"B ");
        }

        stat = hipblasCreate(&handle);

        start = chrono::high_resolution_clock::now(); //start time measure

        //----------------------CUBLAS CHARGE CODE----------------------

        stat = hipblasSetMatrix(dim,dim,data_size,A,dim,gpu_A,dim);//a -> gpu_A
        stat = hipblasSetMatrix(dim,dim,data_size,B,dim,gpu_B,dim);//b -> gpu_B
        stat = hipblasSetMatrix(dim,dim,data_size,C,dim,gpu_C,dim);//c -> gpu_C

        hipDeviceSynchronize();

        //----------------------CUBLAS CHARGE CODE----------------------

        finish = chrono::high_resolution_clock::now(); //end time measure
        elapsed = finish - start; //compute time difference

        cout << "MUL_GCHR: With dimension " << dim << ", elapsed time: " << elapsed.count() << " s" << endl;
        
        hipEventRecord(begin, 0);
        
        //----------------------CUBLAS PARALLEL CODE----------------------

        // C := alfa*A*B + beta*C;
        stat=hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,dim,dim,dim,&alfa,gpu_A,dim,gpu_B,dim,&beta,gpu_C,dim);

        //----------------------CUBLAS PARALLEL CODE---------------------- 

        hipDeviceSynchronize(); //to reassure everything is in sync

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &time, begin, stop);        

        cout << "MUL_PRLL: With dimension " << dim << ", elapsed time: " << time << " ms" << endl;

        start = chrono::high_resolution_clock::now(); //start time measure

        //----------------------CUBLAS DISCHARGE CODE----------------------
        //si basa sul fatto che i caricamenti sono sincroni, mentre l'esecuzione parallela no        

        stat=hipblasGetMatrix(dim,dim,data_size,gpu_C,dim,C,dim); // gpu_C -> C

        hipDeviceSynchronize();

        //----------------------CUBLAS DISCHARGE CODE----------------------

        finish = chrono::high_resolution_clock::now(); //end time measure

        elapsed = finish - start; //compute time difference

       cout << "MUL_CCHR: With dimension " << dim << ", elapsed time: " << elapsed.count() << " s" << endl;

        if(DEBUG){
            print_array_as_matrix(C,dim,"MULT ");
        }

        hipFree(gpu_A);
        hipFree(gpu_B);
        hipFree(gpu_C);
        free(B);
        free(C);
        hipblasDestroy(handle);   

        B = new float[dim];
        C = new float[dim];

        for(int i=0;i<dim;i++) B[i] = 0.0;                //  initialize B
        for(int i=0;i<dim;i++) C[i] = 1.0;    // C - N-vector  of ones

        cusolverStatus = hipsolverDnCreate (& cuhandle ); 

        status = hipMalloc((void**) &gpu_A, data_size);
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        status = hipMalloc((void**) &gpu_B, dim*sizeof(float));  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        status = hipMalloc((void**) &gpu_pivot, dim*sizeof(int));  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        status = hipMalloc((void**) &gpu_info, dim*sizeof(int));  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        smaller_size = dim*sizeof(float);

        start = chrono::high_resolution_clock::now(); //start time measure

        //----------------------CUBLAS CHARGE CODE----------------------
        //si basa sul fatto che i caricamenti sono sincroni, mentre l'esecuzione parallela no        

        status = hipMemcpy(gpu_A, A, data_size,hipMemcpyHostToDevice);      // copy d_A <-A

        //moltiplica B = A*C
        cblas_sgemv(CblasColMajor,CblasNoTrans,dim,dim,alfa,A,dim,C,incx,beta,B,incy);

        status = hipMemcpy(gpu_B, B, smaller_size,hipMemcpyHostToDevice);      // copy d_B <-B

        cusolverStatus = hipsolverDnSgetrf_bufferSize(cuhandle,dim,dim,gpu_A,dim,&Lwork);      //  compute  buffer  size  and  prep.memory

        hipDeviceSynchronize();

        //----------------------CUBLAS CHARGE CODE----------------------

        finish = chrono::high_resolution_clock::now(); //end time measure
        elapsed = finish - start; //compute time difference
        cout << "INV_GCHR: With dimension " << dim << ", elapsed time: " << elapsed.count() << " s" << endl;
        

        status = hipMalloc((void**) &gpu_Work, Lwork*sizeof(float));  
        
        if(status!=hipSuccess){
            cout << hipGetErrorString(status) << " in " << __FILE__ << " at line " << __LINE__ << endl;
        }

        hipEventRecord(begin, 0);

        //----------------------CUBLAS PARALLEL CODE----------------------

        cusolverStatus = hipsolverDnSgetrf(cuhandle,dim,dim,gpu_A,dim,gpu_Work,gpu_pivot,gpu_info);
        cusolverStatus = hipsolverDnSgetrs(cuhandle, HIPBLAS_OP_N,dim,1,gpu_A,dim,gpu_pivot,gpu_B,dim,gpu_info);

        //----------------------CUBLAS PARALLEL CODE---------------------- 

        hipDeviceSynchronize(); //to reassure everything is in sync

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &time, begin, stop);

        cout << "INV_PRLL: With dimension " << dim << ", elapsed time: " << time << " ms" << endl;
      
        start = chrono::high_resolution_clock::now(); //start time measure

        //----------------------CUBLAS DISCHARGE CODE----------------------
        //si basa sul fatto che i caricamenti sono sincroni, mentre l'esecuzione parallela no        

        status = hipMemcpy (&info_gpu , gpu_info , sizeof(int), hipMemcpyDeviceToHost );
        if(DEBUG) cout << "after getrf+getrs: info_gpu = " << info_gpu << endl;
        status = hipMemcpy(C, gpu_B , dim*sizeof(float), hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        //----------------------CUBLAS DISCHARGE CODE----------------------

        finish = chrono::high_resolution_clock::now(); //end time measure

        elapsed = finish - start; //compute time difference
         
        cout << "INV_CCHR: With dimension " << dim << ", elapsed time: " << elapsed.count() << " s" << endl;
         
        if(DEBUG){
            print_array_as_matrix(C,dim,"C ");
        }

        //deallocate things

        hipFree(gpu_A);
        hipFree(gpu_B);
        hipFree(gpu_pivot);
        hipFree(gpu_info);
        hipFree(gpu_Work);
        free(A);
        free(B);
        free(C);
        cusolverStatus = hipsolverDnDestroy(cuhandle);       
        status = hipDeviceReset();
    }
 
    hipEventDestroy(begin);
    hipEventDestroy(stop);

    return 0;
}